#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
//#include "../common/book.h"
//#include "../common/cpu_bitmap.h"

#define NN (640 * 480)//(33*1024)
#define length (640 * 480)
#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))  
#define MAX(X,Y) ((X) > (Y) ? (X) : (Y))  
hipError_t convWithCuda(double *b, const double *a, const int width, const int height, const int fwidth);
hipError_t smoothingWithCuda(double *b, const double *a, const double filter_size, const int size);
hipError_t normWithCuda(double *b, const double *a, const double min, const double max, const int size);
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__global__ void smoothingKernel(double *b, const double *a, double *f, const double filter_size, const double filter_sum, const int size)
{
	double value = 0;
	int filter_pad = filter_size / 2;
	int halfy = filter_pad;
	int halfx = filter_pad;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int x = tid % 640;
	int y = tid / 640;
//while (tid >= filter_pad && tid < size-filter_pad) {
//	double value = 0;
		//for (int n = -halfy; n <= halfy; n++) 
		//{
		//	for (int m = -halfx; m <= halfx; m++)
		//	{
		//		value += (double)a[   ( ( ( y+n ) * 640) + x + m ) ] *      f[ (n + halfy) * (int)(filter_size)    + (m + halfx) ];
		//		//value += (double)a[ ( ( ( y+n ) * 20 ) + x + m ) ] * matrix[ (n + halfy) * (int)filter_size + (m + halfx) ];
		//	}
		//}
		//value = value / filter_sum;
		//b[tid] = value;
		b[tid] = a[tid];
//	tid += blockDim.x * gridDim.x;
//}
}

__global__ void normKernel(double *b, const double *a, const double min, const double max, const int size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < size) {
		b[tid] = (a[tid] - min) / (max - min) * 255;
		tid += blockDim.x * gridDim.x;
	}
}


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

extern "C" __declspec(dllexport) void exportCppFunctionConv(double* dst, double* src, int width, int height, int fwidth)
{
	//	printf("exportCppFunctionNorm \n");

	hipError_t cudaStatus = convWithCuda(dst, src, width, height, fwidth);
	if (cudaStatus != hipSuccess) fprintf(stderr, "smoothingWithCuda failed!");
	//else printf("exportCppFunctionNorm Success \n");
}

extern "C" __declspec(dllexport) void exportCppFunctionSmoothing(double* dst, double* src, double filter_size, int arraySize)
{
	//	printf("exportCppFunctionNorm \n");

	hipError_t cudaStatus = smoothingWithCuda(dst, src, filter_size, arraySize);
	if (cudaStatus != hipSuccess) fprintf(stderr, "smoothingWithCuda failed!");
	//else printf("exportCppFunctionNorm Success \n");
}


extern "C" __declspec(dllexport) void exportCppFunctionNorm(double* dst, double* src, double min, double max, int arraySize)
{
//	printf("exportCppFunctionNorm \n");

	hipError_t cudaStatus = normWithCuda(dst, src, min, max, arraySize);
	if (cudaStatus != hipSuccess) fprintf(stderr, "normWithCuda failed!");
	//else printf("exportCppFunctionNorm Success \n");
}


extern "C" __declspec(dllexport) void exportCppFunctionAdd(int* src, int* src2, int* dst, int arraySize)
{
	printf("exportCppFunctionAdd \n");

	hipError_t cudaStatus = addWithCuda(dst, src, src2, arraySize);
	if (cudaStatus != hipSuccess) fprintf(stderr, "addWithCuda failed!");
	else printf("exportCppFunctionAdd Success \n");
}
__global__ void VectorAdd(const double*a, const double*b, double*c, double size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	c[tid] = a[tid] + b[tid];
}
__global__ void add(double *a, double *b, double *c)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//while (tid < N) {
		c[tid] = a[tid] + b[tid];
	//	tid += blockDim.x * gridDim.x;
	//}
}
__global__ void MatrixConv(double*P, double*M, double*N,  int Width, int Height, int fWidth, double fsum)
{
	int tid, tx, ty;
	//2���� �۾� ���� �ε��� ���
	tx = blockDim.x * blockIdx.x + threadIdx.x;
	ty = blockDim.y * blockIdx.y + threadIdx.y;
	tid = Width * ty + tx;
	//filter ���� 
	int fPad;
	fPad = fWidth / 2;
	int index_x = 0;
	int index_y = 0;

	double Value = 0;
	double MVal = 0;
	double NVal = 0;

	if (tx >= fPad && tx < Width - fPad) {
		if (ty >= fPad && ty < Height - fPad) {
			for (index_y = -fPad; index_y <= fPad; index_y++) { //����ũ�⸸ŭ ��ȯ
				for (index_x = -fPad; index_x <= fPad; index_x++)
				{
					int fcol = ty + index_y;
					int frow = tx + index_x;
					int findex = fcol*Width + frow;
					int findex_y = index_y + fPad;
					int findex_x = index_x + fPad;
					int index = findex_y * fWidth + findex_x;
					MVal = M[findex];
					NVal = N[index];
					Value += MVal * NVal;
				}
			}
			P[tid] = Value / 9.0;
		}
	}
}
void MatrixConvC(int*M, int*N, int*P, int Width, int Height, int fWidth) {
	int col = 0;
	int raw = 0;
	int index_x = 0;
	int index_y = 0;
	int Destindex = 0;

	//filter ���� 
	int fPad;
	fPad = fWidth / 2;

	for (col = fPad; col < Height - fPad; col++) {
		for (raw = fPad; raw < Width - fPad; raw++) {
			Destindex = col*Width + raw;
			for (index_y = -fPad; index_y <= fPad; index_y++) {
				for (index_x = -fPad; index_x <= fPad; index_x++) {
					int fcol = col + index_y;
					int frow = raw + index_x;
					int findex = fcol*Width + frow;
					int findex_y = index_y + fPad;
					int findex_x = index_x + fPad;
					int index = findex_y * fWidth + findex_x;
					P[Destindex] += M[findex] * N[index];
				}
			}
			P[Destindex] = P[Destindex] / 9;
		}
	}
}
int main()
{
	//const int MatrixWidth = 640;
	//const int MatrixHeight = 480;
	//const int MatrixSize = MatrixWidth*MatrixHeight;
	//const int BufferSize1 = MatrixSize * sizeof(int);

	//const int FilterWidth = 5;
	//const int FilterHeight = 5;
	//const int FilterSize = FilterWidth*FilterHeight;
	//const int BufferSize2 = FilterSize * sizeof(int);

	//int* M;
	//int* N;
	//int* P_cuda;
	//int* P_C;

	////ȣ��Ʈ �޸� �Ҵ�
	//M = (int*)malloc(BufferSize1);
	//N = (int*)malloc(BufferSize2);
	//P_cuda = (int*)malloc(BufferSize1);
	//P_C = (int*)malloc(BufferSize1);

	//int i, j = 0;

	////������ �Է�
	//for (int i = 0; i < MatrixSize; i++) {
	//	M[i] = i;
	//	//N[i] = i;
	//	P_cuda[i] = 0;
	//	P_C[i] = 0;
	//}
	//for (int i = 0; i < FilterSize; i++) {
	//	N[i] = 1;
	//}

	//int* dev_M;
	//int* dev_N;
	//int* dev_P;

	////����̽� �޸� �Ҵ�
	//hipMalloc((void**)&dev_M, BufferSize1);
	//hipMalloc((void**)&dev_N, BufferSize2);
	//hipMalloc((void**)&dev_P, BufferSize1);

	////ȣ��Ʈ ����̽� �Է� ������ ����
	//hipMemcpy(dev_M, M, BufferSize1, hipMemcpyHostToDevice);
	//hipMemcpy(dev_N, N, BufferSize2, hipMemcpyHostToDevice);

	////dim3 Dg(3, 4, 1);
	//dim3 Dg(20, 32, 1);
	////dim3 Db(20, 32, 1);
	////dim3 Db(4, 3, 1);
	//dim3 Db(32, 15, 1);

	////CUDA kernel ��Ʈ���� �� ���
	//MatrixConv << <Dg, Db >> > (dev_P, dev_M, dev_N,  MatrixWidth, MatrixHeight, FilterWidth);

	////����̽� ȣ��Ʈ ��� ������ ����
	//hipMemcpy(P_cuda, dev_P, BufferSize1, hipMemcpyDeviceToHost);

	////C �Լ� ��Ʈ���� �� ���
	//MatrixConvC(M, N, P_C, MatrixWidth, MatrixHeight, FilterWidth);

	//bool ResultFlag = true;
	////��� ���
	//for (i = 0; i < MatrixSize; i++) {
	//	//printf("Result[%d] : %d, %d\n",i,P_cuda[i], P_C[i]);
	//	if (P_cuda[i] != P_C[i]) ResultFlag = false;
	//}
	//for (i = 0; i < 12; i++) {
	//	for (j = 0; j < 12; j++) {
	//		printf("Result[%d, %d] : %d, %d\n", i, j, P_cuda[i*MatrixHeight + j], P_C[i*MatrixHeight + j]);
	//		//if (P_cuda[i] != P_C[i]) ResultFlag = false;
	//	}
	//}
	////MatrixWidth MatrixHeight

	//if (ResultFlag == true) printf("MatrixMul Result OK!\n");
	//else printf("MatrixMul Result Error!\n");

	//hipFree(dev_M);
	//hipFree(dev_N);
	//hipFree(dev_P);

	//free(M);
	//free(N);
	//free(P_cuda);
	//free(P_C);

	return 0;
}
//int main()
//{
//	//int a[N], b[N], c[N];
//	double* a; a = (double*)malloc(N * sizeof(double));
//	double* b; b = (double*)malloc(N * sizeof(double));
//	double* c; c = (double*)malloc(N * sizeof(double));
//
//	double *dev_a, *dev_b, *dev_c;
//
//	//GPU �޸𸮸� �Ҵ��Ѵ�.
//	hipMalloc((void**)&dev_a, N * sizeof(double));
//	hipMalloc((void**)&dev_b, N * sizeof(double));
//	hipMalloc((void**)&dev_c, N * sizeof(double));
//
//	//CPU�� �迭 'a'�� 'b'�� ä���.
//	for (int i = 0; i < N; i++) {
//		a[i] = (double)i;
//		b[i] = (double)i * (double)i;
//	}
//
//	//�迭 'a'�� 'b'�� GPU�� �����Ѵ�.
//	hipMemcpy(dev_a, a, N * sizeof(double), hipMemcpyHostToDevice);
//	hipMemcpy(dev_b, b, N * sizeof(double), hipMemcpyHostToDevice);
//
//	/////////////////////////////////////////////////////////////////////////////////////////////////////
//	//add << <640, 480 >> > (dev_a, dev_b, dev_c);
//	hipError_t cudaStatus = smoothingWithCuda(b, a, 41, N);
//	/////////////////////////////////////////////////////////////////////////////////////////////////////
//	//�迭 'c'�� GPU���� �ٽ� CPU�� �����Ѵ�.
//	hipMemcpy(c, dev_c, N * sizeof(double), hipMemcpyDeviceToHost);
//
//	//�츮�� ��û�� �۾��� GPU�� �����Ͽ����� Ȯ���Ѵ�.
//	bool success = true;
//	for (int i = 0; i < N; i++) {
//		if (a[i] != b[i] ) {//(a[i] + b[i] != c[i]) {
//			printf("Error: %d + %d != %d\n", a[i], b[i], c[i]);
//			success = false;
//		}
//	}
//	if (success) printf("We did it!\n");
//
//	//GPU�� �Ҵ��� �޸𸮸� �����Ѵ�.
//	free(a);
//	free(b);
//	free(c);
//	hipFree(dev_a);
//	hipFree(dev_b);
//	hipFree(dev_c);
//	/////////////////////////////////////////////////////////////////////////////////////////////////////
//	//hipError_t cudaStatus = smoothingWithCuda(b, a, filter_size, arraySize);
//	/////////////////////////////////////////////////////////////////////////////////////////////////////
//	//const double size = 640 * 480;
//	//const double BufferSize = size * sizeof(double);
//
//	//double* InputA;
//	//double* InputB;
//	//double* Result;
//
//	////ȣ��Ʈ �޸� �Ҵ�
//
//	//InputA = (double*)malloc(BufferSize);
//	//InputB = (double*)malloc(BufferSize);
//	//Result = (double*)malloc(BufferSize);
//
//	//int i = 0;
//
//	////������ �Է�
//	//for (int i = 0; i < size; i++)
//	//{
//	//	InputA[i] = i;
//	//	InputB[i] = i;
//	//	Result[i] = 0;
//	//}
//
//	//double* dev_A;
//	//double* dev_B;
//	//double* dev_R;
//
//	////����̽� �޸� �Ҵ�
//	//hipMalloc((void**)&dev_A, size * sizeof(double));
//	//hipMalloc((void**)&dev_B, size * sizeof(double));
//	//hipMalloc((void**)&dev_R, size * sizeof(double));
//
//	////ȣ��Ʈ ����̽� �Է� ������ ����
//	//hipMemcpy(dev_A, InputA, size * sizeof(double), hipMemcpyHostToDevice);
//	//hipMemcpy(dev_B, InputB, size * sizeof(double), hipMemcpyHostToDevice);
//
//	////33,553,920 ���� �����带 �����Ͽ� ���� ���
//	//VectorAdd << <640, 480 >> > (dev_A, dev_B, dev_R, size);
//	////����̽� ȣ��Ʈ ��� ������ ����
//	//hipMemcpy(Result, dev_R, size * sizeof(double), hipMemcpyDeviceToHost);
//
//	////��� ���
//	//for (i = 0; i < 5; i++) {
//	//	printf(" Result[%d] : %f\n", i, Result[i]);
//	//}
//	//printf("......\n");
//	//for (i = size - 5; i < size; i++) {
//	//	printf(" Result[%d] : %f\n", i, Result[i]);
//	//}
//	////����̽� �޸� ����
//	//hipFree(dev_A);
//	//hipFree(dev_B);
//	//hipFree(dev_R);
//
//	////ȣ��Ʈ �޸� ����
//	//free(InputA);
//	//free(InputB);
//	//free(Result);
//	/////////////////////////////////////////////////////////////////////////////////////////////////////
//	////const double size = 640 * 480;
//	////const double BufferSize = size * sizeof(double);
//
//	//double* a;
//	////double* InputB;
//	//double* b;
//
//	//////ȣ��Ʈ �޸� �Ҵ�
//
//	//a = (double*)malloc(length * sizeof(double));
//	////InputB = (double*)malloc(BufferSize);
//	//b = (double*)malloc(length * sizeof(double));
//	///////////////////
// //   //double LUT1D[length], PDFLUT[length];
//	//int arraySize = sizeof(a) / sizeof(double);
//	//int max = 255;
//	//int min = 0;
//
//	////int a[length], b[length], c[length];
//	//for (int i = 0; i < length; i++) {
//	//	a[i] = (double)i;
//	//	b[i] = (double)0;
//	//}
//	//int filter_size = 21;
//	//int f_size = filter_size;
//	//int total_size = f_size * f_size;
//	//double sigma = 2.0;
//	//double* matrix = new double[total_size];
//	////for (int x = 0; x < f_size; x++)
//	////	matrix[x] = new double[f_size];
//	//int halfx = f_size / 2;
//	//int halfy = f_size / 2;
//	//double denom = -2.0 * pow(sigma, 2);
//	//double sum = 0.0;
//	//for (int y = -halfy; y <= halfy; y++) {
//	//	int i = y + halfy;
//	//	for (int x = -halfx; x <= halfx; x++) {
//	//		int j = x + halfx;
//	//		//Gaussian �Լ� �� ����ϱ�
//	//		double value = (double)exp((double)(x*x + y*y) / denom);
//	//		matrix[(i*f_size) + j] = value;
//	//		sum += value;
//	//		printf("%f ", matrix[(i*f_size) + j]);
//	//	}
//	//	printf("\n");
//	//}
//
//	//int x = 641 % 640;
//	//int y = 641 / 640;
//	//printf("\n\n\n\n");
//	//int filter_pad = filter_size / 2;
//	//int size = length;
//	//for (int y = filter_pad; y < 480- filter_pad; y++) {
//	//	for (int x = filter_pad; x < 640 - filter_pad; x++) {
//	//		double value = 0;
//	//		for (int n = -halfy; n <= halfy; n++) {
//	//			int i = n + halfy;
//	//			for (int m = -halfx; m <= halfx; m++) {
//	//				int j = m + halfx;
//	//				value += (double)a[ (((y + n) * 20) + x + m) ] * matrix[ i * f_size + j ];
//	//				//printf("%f ", value);
//	//			}
//	//			//printf("\n");
//	//		}
//	//		value = value / sum;
//	//		b[y * filter_size + x] = value;
//	//	}
//	//}
//	//int aMax = 0;
//	//int bMax = 0;
//	//for (int y = 0; y < 480; y++) {
//	//	for (int x = 0; x < 640; x++) {
//	//		//min(1, 1);
//	//		aMax = MAX(aMax, a[y * 20 + x]);
//	//		bMax = MAX(bMax, b[y * 20 + x]);
//	//	}
//	//}
//	//
// //   // Add vectors in parallel.
// //   hipError_t cudaStatus = smoothingWithCuda(b, a, filter_size, arraySize);
//	////hipError_t cudaStatus = normWithCuda(PDFLUT, LUT1D, min, max, arraySize);
//	////hipError_t cudaStatus = addWithCuda(c, a, b, length);
// //   if (cudaStatus != hipSuccess) {
// //       fprintf(stderr, "addWithCuda failed!");
// //       return 1;
// //   }
//	//
// //   // hipDeviceReset must be called before exiting in order for profiling and
// //   // tracing tools such as Nsight and Visual Profiler to show complete traces.
// //   cudaStatus = hipDeviceReset();
// //   if (cudaStatus != hipSuccess) {
// //       fprintf(stderr, "hipDeviceReset failed!");
// //       return 1;
// //   }
//
//    return 0;
//}
hipError_t convWithCuda(double *b, const double *a, const int Width, const int Height, const int fWidth)
{
	double *dev_a = 0;
	double *dev_b = 0;
	double *dev_f = 0;
	hipError_t cudaStatus;
	/////////////////////////////////////////////////////////////////////////////////
	//image size
	int size = Width * Height;
	//Prepare to filter
	double f_size = fWidth;
	double total_size = f_size * f_size;
	double sigma = 4.0;
	double* matrix = new double[(int)total_size];
	//double* matrix = (double*)malloc((int)f_size * sizeof(double));
	//for (int x = 0; x < f_size; x++)
	//	matrix[x] = new double[f_size];
	int halfx = (int)f_size / 2;
	int halfy = (int)f_size / 2;
	double denom = -2.0 * pow(sigma, 2);
	double sum = 0.0;
	for (int y = -halfy; y <= halfy; y++) {
		int i = y + halfy;
		for (int x = -halfx; x <= halfx; x++) {
			int j = x + halfx;
			//Gaussian �Լ� �� ����ϱ�
			double value = (double)exp((double)(x*x + y*y) / denom); //1/pow(filter_size,2);//
			matrix[(i*(int)f_size) + j] = value;
			sum += value;
		}
	}
	/////////////////////////////////////////////////////////////////////////////////
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_f, total_size * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_f, matrix, total_size * sizeof(double), hipMemcpyHostToDevice);////////////////////////////////////
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	//smoothingKernel << <640, 480 >> >(dev_b, dev_a, dev_f, f_size, sum, size);

	dim3 Dg(20, 32, 1);
	dim3 Db(32, 15, 1);
	//CUDA kernel ��Ʈ���� �� ���
	MatrixConv << <Dg, Db >> > (dev_b, dev_a, dev_f, Width, Height, fWidth, sum);
	//smoothingKernel1(dev_b, dev_a, dev_f, halfx, sum, size);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "smoothingKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching smoothingKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(b, dev_b, size * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_f);
	//free(matrix);
	delete[] matrix;
	return cudaStatus;
}
hipError_t smoothingWithCuda(double *b, const double *a, const double filter_size, const int size)
{
	double *dev_a = 0;
	double *dev_b = 0;
	double *dev_f = 0;
	hipError_t cudaStatus;
	/////////////////////////////////////////////////////////////////////////////////
	//Prepare to filter
	double f_size = filter_size;
	double total_size = f_size * f_size;
	double sigma = 4.0;
	double* matrix = new double[(int)total_size];
	//double* matrix = (double*)malloc((int)f_size * sizeof(double));
	//for (int x = 0; x < f_size; x++)
	//	matrix[x] = new double[f_size];
	int halfx = (int)f_size / 2;
	int halfy = (int)f_size / 2;
	double denom = -2.0 * pow(sigma, 2);
	double sum = 0.0;
	for (int y = -halfy; y <= halfy; y++) {
		int i = y + halfy;
		for (int x = -halfx; x <= halfx; x++) {
			int j = x + halfx;
			//Gaussian �Լ� �� ����ϱ�
			double value = (double)exp((double)(x*x + y*y) / denom); //1/pow(filter_size,2);//
			matrix[(i*(int)f_size) + j] = value;
			sum += value;
		}
	}
	/////////////////////////////////////////////////////////////////////////////////
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_f, total_size * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_f, matrix, total_size * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	smoothingKernel << <640, 480 >> >(dev_b, dev_a, dev_f, f_size, sum, size);
	//smoothingKernel1(dev_b, dev_a, dev_f, halfx, sum, size);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "smoothingKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching smoothingKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(b, dev_b, size * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_f);
	//free(matrix);
	delete[] matrix;
	return cudaStatus;
}



hipError_t normWithCuda(double *b, const double *a, const double min, const double max, const int size)
{
	double *dev_a = 0;
	double *dev_b = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	normKernel << <640, 480 >> >(dev_b, dev_a, min, max, size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "normKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching normKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(b, dev_b, size * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
